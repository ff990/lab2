
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512


__global__ void naiveReduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    // NAIVE REDUCTION IMPLEMENTATION
    
    __shared__ float sum[2*BLOCK_SIZE];
    unsigned int a=threadIdx.x;
    unsigned int i=2*blockIdx.x*blockDim.x;
    
    sum[a]=in[i+a];
    sum[blockDim.x+a]=in[i+blockDim.x+a];
    for(unsigned int j=1;j<=blockDim.x;j*=2)
    {
        __syncthreads();
        if(a%j==0)
            sum[2*a]+=sum[2*a+j];
    }
    if(a==0)
        out[blockIdx.x]=sum[0];

}

__global__ void optimizedReduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    // OPTIMIZED REDUCTION IMPLEMENTATION
    
    __shared__ float sum[2*BLOCK_SIZE];
    unsigned int a=threadIdx.x;
    unsigned int i=2*blockIdx.x*blockDim.x;
    
    if(a+i<size)
        sum[a]=in[a+i];
    else
        sum[a]=0.0;
    
    if(blockDim.x+a+i<size)
        sum[blockDim.x+a]=in[blockDim.x+a+i];
    else
        sum[blockDim.x+a]=0.0;
    
    for(unsigned int j=blockDim.x;j>0;j/=2)
    {
         __syncthreads();
        if(a<j)
            sum[a]+=sum[a+j];
    }
    if(a==0)
        out[blockIdx.x]=sum[0];


}

